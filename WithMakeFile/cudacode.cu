//#include <cuda>

//#include <cuda.h>
//#include <cuda_runtime.h>
//#include <cuda_runtime_api.h>


#include <hip/hip_runtime.h>
#include "stdio.h"

__global__
void testKernel(float* d_data)
{
	int myId = threadIdx.x;
	d_data[myId] = 10;
}

void CallKernel(int a, int b)
{
	int threads = 32;

	dim3 gridSize(1, 1, 1);
	dim3 blockSize(threads, 1, 1);

	float* h_data;
	float* d_data;
	int dataLen = threads;

	h_data = (float *)malloc(sizeof(float) * dataLen);

	for (int i = 0; i < dataLen; i++)
	{
		h_data[i] = i;
	}

	hipMalloc((void**)&d_data, sizeof(float) * dataLen);
	hipMemcpy(d_data, h_data, sizeof(float) * dataLen, hipMemcpyHostToDevice);

	puts("starting kernel");
	testKernel<<<gridSize, blockSize>>>(d_data);
	puts("kernel finished");

	hipMemcpy(h_data, d_data, sizeof(float) * dataLen, hipMemcpyDeviceToHost);

	puts("data:");
	for (int i = 0; i < dataLen; i++)
	{
		printf(" %f\n", h_data[i]);
	}
	puts("end");

	free(h_data);
	hipFree(d_data);
	hipError_t error = hipGetLastError();
  	if(error != hipSuccess)
   	{
    	// print the CUDA error message and exit
    	printf("CUDA error: %s\n", hipGetErrorString(error));
    	exit(-1);
  	}
}

/*
int main(int argc, char* argv[])
{
	int a = 1;
	int b = 2;
	CallKernel(a, b);
}
*/